#include "hip/hip_runtime.h"
#include <__clang_cuda_builtin_vars.h>
#include <__clang_cuda_runtime_wrapper.h>
#include <cstdlib>
#include <hip/hip_runtime.h>
#define M 10000
#define N 10000
#define K 10000


__global__
void naive_matmul(
    float* matrixA,
    float* matirxB,
    float* output
){
    int global_index_x = blockDim.x * blockIdx.x + threadIdx.x;
    // int global_index_y = blockDim.y * blockIdx.y + threadIdx.y;
    int x = global_index_x  / M;
    int y = global_index_x % N;

    if(x < M && y < N){
        for(int k = 0 ; k < K ; k++){
            output[x * M + y] = matrixA[x*K +k] * matirxB[k*N + y];
        }
    }
    
}


void launch_naive_matmul(
    float* matrixA,
    float* matrixB,
    float* output
){

    dim3 blockDim()

}







float time_it(void (*function)(float* , float* , float*) ,float* matrixA , float* matrixB , float* output){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    //launch kernel
    function(matrixA , matrixB , output);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop); 
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

}


int main(){

    float* matrixA_host = new float[M*K];
    float* matrixB_host = new float[K*N];
    float* matrix_output_host = new float[M*N];

    float* matrixA , *matrixB , *matrixO;
    hipMalloc(&matrixA , M*K*sizeof(float));
    hipMalloc(&matrixB , N*K*sizeof(float));
    hipMalloc(&matrixO , N*M*sizeof(float));

    //init
    for(int i = 0 ; i < M*K ; i++){
        matrixA_host[i] = rand();
    }
    for(int i = 0 ; i < N*K ; i++){
        matrixB_host[i] = rand();
    }

    //copy
    hipMemcpy(matrixA , matrixA_host , M*K*sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy(matrixB , matrixB_host , N*K*sizeof(float) , hipMemcpyHostToDevice);

    



    
}