#include <__clang_cuda_builtin_vars.h>
#include <__clang_cuda_runtime_wrapper.h>
#include <cmath>
#include <iostream>

#include <cstddef>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <numeric>


__global__
void findSum(
    float* vector,
    float* output,
    int N
){
    int global_index = blockIdx.x * blockDim.x + threadIdx.x;
    int local_index = threadIdx.x;
    extern __shared__ float SHARED_MEM[];

    if(local_index < N){
        SHARED_MEM[local_index] = vector[global_index];
    }
    else{
        SHARED_MEM[local_index] = 0;
    }

    for(int stride = 1 ; stride < blockDim.x ; stride *= 2){

        if(local_index % (2 * stride) == 0){
            SHARED_MEM[local_index] += SHARED_MEM[local_index + stride];
        }

        __syncthreads();
    }

    if(local_index == 0){
        output[blockIdx.x] = SHARED_MEM[0];
    }

}




float launch_findSum(
    float *vector,
    int N
){
    int threadsPerBlock = 256;
    int blockPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    size_t shared_mem_size = threadsPerBlock * sizeof(float);
    float* output;
    hipMalloc((void**)&output , sizeof(float) * N);
    findSum<<<
        blockPerGrid,
        threadsPerBlock,
        shared_mem_size
        >>>(
            vector,
            output,
            N
        );

    if(blockPerGrid > 1){
        findSum<<<
            blockPerGrid,
            threadsPerBlock,
            shared_mem_size
            >>>(output, output, blockPerGrid);
    }


    float* h_output;
    h_output = new float[blockPerGrid];
    hipMemcpy(h_output , output , blockPerGrid * sizeof(float) , hipMemcpyDeviceToHost);

    // std::cout << h_output[0] << std::endl;
    return h_output[0];
}



//----------------------------


__global__
void warpSum(
    float* vector,
    float* output,
    int N
){
    int laneId = threadIdx.x % 32;

    float value = vector[threadIdx.x];
    for(int offset = 16 ; offset > 0 ; offset /=2){
        value += __shfl_xor_sync(0xFFFFFFFF , value , offset , 32);
    }

    if(laneId == 0){
        output[threadIdx.x / 32] = value;
    }

}



int main(){
    int N = 1000;

    float* input;
    input = new float[N];
    
    //
    int sum_verify = 0;
    for(int i = 0 ; i < N ; i++){
        input[i] = i;
        sum_verify += i;
    }

    float* vector;
    hipMalloc((void**)&vector , sizeof(float)* N);
    hipMemcpy(vector , input , N * sizeof(float) , hipMemcpyHostToDevice);

    float naive_out = launch_findSum(vector,  N);

    // std::cout << std::accumulate(input , input + N ,0) << std::endl;
    std::cout << (sum_verify == naive_out) << std::endl;

    
}